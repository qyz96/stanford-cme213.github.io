#include "hip/hip_runtime.h"
/* This is machine problem 1, part 1, shift problem
 *
 * The problem is to take in a string (a vector of characters) and a shift amount,
 * and add that number to each element of
 * the string, effectively "shifting" each element in the
 * string.
 *
 * We do this in three different ways:
 * 1. With a cuda kernel loading chars and outputting chars for each thread
 * 2. With a cuda kernel, casting the character pointer to an int so that
 *    we load and store 4 bytes each time instead of 1 which gives us better coalescing
 *    and uses the memory effectively to achieve higher bandwidth
 * 3. Same spiel except with a uint2, so that we load 8 bytes each time
 *
 */

 #include <algorithm>
 #include <cstdlib>
 #include <iostream>
 #include <iomanip>
 #include <ctime>
 #include <fstream>
 #include <vector>
 
 #include "util.cuh"
 #include "shift.cuh"
 
 
 constexpr const char *MOBY_DICK = "mobydick.txt";
 constexpr const int CUDA_BLOCK_SIZE = 256;
 
 void host_shift(std::vector<unsigned char> &input_array, 
                 std::vector<unsigned char> &output_array,
                 unsigned char shift_amount) 
 {
     std::transform(input_array.begin(), input_array.end(), output_array.begin(),
         [&shift_amount](unsigned char& element) {
             return element + shift_amount;
         }
     );
 }
 
 void checkResults(std::vector<unsigned char> &text_host, 
                   unsigned char *device_output_array,
                   unsigned int num_entries, 
                   const char *type) 
 {
     // allocate space on host for gpu results
     std::vector<unsigned char> text_from_gpu(num_entries);
 
     // download and inspect the result on the host:
     hipMemcpy(&text_from_gpu[0], device_output_array, num_entries,
                hipMemcpyDeviceToHost);
     check_launch("copy from gpu");
 
     // check CUDA output versus reference output
     int error = 0;
 
     for (unsigned int i = 0; i < num_entries; i++) 
     {
         if (text_host[i] == text_from_gpu[i])
             continue;
 
         ++error;
         std::cerr << "Mismatch at pos: " << i << std::endl
                   << "Expected " << static_cast<int>(text_host[i])
                   << " and got " << static_cast<int>(text_from_gpu[i])
                   << std::endl;
 
         if (error > 10) 
         {
             std::cerr << std::endl << "Too many errors, quitting..." << std::endl;
             break;
         }
     }
 
     if (error) 
     {
         std::cerr << "\nError(s) in " << type << " kernel!" << std::endl;
         exit(1);
     }
 }
 
 int main(int argc, char** argv) 
 {
     int exit_code = 0;
     
     // check that the correct number of command line arguments were given
     if (argc != 2)
     {
         std::cerr << "usage: " << argv[0] << " <int>" << std::endl
                   << "Must supply the number of times to double the input file!" 
                   << std::endl;
         return 1;
     }
 
     // convert argument to integer
     int number_of_doubles = std::atoi(argv[1]);
     if (number_of_doubles < 0)
     {
         std::cerr << "usage: " << argv[0] << " <int>" << std::endl
                   << "Number of time times must be at least zero." 
                   << std::endl;
         return 1;
     }
 
     hipFree(0); //initialize cuda context to avoid including cost in timings later
 
     // Warm-up each of the kernels to avoid including overhead in timing.
     // If the kernels are written correctly, then they should
     // never make a bad memory access, even though we are passing in NULL
     // pointers since we are also passing in a size of 0
     shift_char <<<1, 1>>>(nullptr, nullptr, 0, 0);
     shift_int  <<<1, 1>>>(nullptr, nullptr, 0, 0);
     shift_int2 <<<1, 1>>>(nullptr, nullptr, 0, 0);
 
     // First load the text
     std::ifstream ifs(MOBY_DICK, std::ios::binary);
     if (!ifs) 
     {
         std::cerr << "Couldn't open " << MOBY_DICK << "!" << std::endl;
         return 1;
     }
 
     std::vector<unsigned char> text;
 
     // get file length by seeking to end of file and getting offset
     // we then re-seek back to the beginning of the file
     ifs.seekg(0, std::ios::end);
     int length = ifs.tellg();
     ifs.seekg(0, std::ios::beg);
 
     // read in the text
     text.resize(length);
     ifs.read((char *) &text[0], length);
     ifs.close();
 
     // make number_of_doubles copies of the text
     // sizes_to_test should hold [1, 2, ...., number_of_doubles] * text.size()
     // text should hould number_of_doubles copies of text
     std::vector<uint> sizes_to_test;
     sizes_to_test.push_back(text.size());
 
     for (int i = 0; i < number_of_doubles; ++i) 
     {
         text.insert(text.end(), text.begin(), text.end());
         sizes_to_test.push_back(text.size());
     }
 
     // allocate host arrays
     std::vector<unsigned char> text_gpu(text.size());
     std::vector<unsigned char> text_host(text.size());
 
     // Compute the size of the arrays in bytes for memory allocation.
     // We need enough padding so that the uint2 access won't be out of bounds.
     const int num_bytes_alloc = (text.size() + 7) * sizeof(unsigned char);
 
     // pointers to device arrays
     unsigned char *device_input_array  = nullptr;
     unsigned char *device_output_array = nullptr;
 
     // hipMalloc device arrays
     hipMalloc((void **) &device_input_array,  num_bytes_alloc);
     hipMalloc((void **) &device_output_array, num_bytes_alloc);
 
     // set the padding to 0 to avoid overflow.
     hipMemset(device_input_array + text.size(), 0, num_bytes_alloc - text.size());
 
     // if either memory allocation failed, report an error message
     if (!device_input_array || !device_output_array) 
     {
         std::cerr << "Couldn't allocate memory!" << std::endl;
         return 1;
     }
 
     // generate random shift in interval [1, 25]
     unsigned char shift_amount = (rand() % 25) + 1;
 
     // Size of text in bytes. This is the largest size that was allocated.
     const int num_bytes = text.size() * sizeof(unsigned char);
 
     // copy input to GPU
     {
         event_pair timer;
         start_timer(&timer);
         hipMemcpy(device_input_array, &text[0], text.size(), hipMemcpyHostToDevice);
         check_launch("copy to gpu");
 
         double elapsed_time_h2d = stop_timer(&timer);
         std::cout << "Host -> Device transfer bandwidth " 
                   << num_bytes / (elapsed_time_h2d / 1000.) / 1E9 
                   << std::endl << std::endl;
     }
 
     // generate reference output
     {
         event_pair timer;
         start_timer(&timer);
         host_shift(text, text_host, shift_amount);
         double elapsed_time_host = stop_timer(&timer);
         std::cout << "Host (reference) solution bandwidth GB/sec: " 
                   << 2 * num_bytes / (elapsed_time_host / 1000.) / 1E9 
                   << std::endl << std::endl;
     }
 
     // CUDA block size
     std::cout << std::setw(45) << "Device Bandwidth GB/sec" << std::endl;
 
     std::cout << std::setw(70) << std::setfill('-') << " " 
               << std::endl << std::setfill(' ');
 
     std::cout << std::setw(15) << " " << std::setw(15) << "char" 
               << std::setw(15) << "uint" << std::setw(15) 
               << "uint2" << std::endl;
               
     std::cout << std::setw(15) << "Problem Size MB" << std::endl;
 
     // Loop through all the problem sizes and generate timing / bandwidth information for each
     // and also check correctness
     for (const uint size_to_test : sizes_to_test) 
     {
         // generate GPU char output
         double elapsed_time_char = doGPUShiftChar(device_input_array,
                                    device_output_array, shift_amount, size_to_test, CUDA_BLOCK_SIZE);
         checkResults(text_host, device_output_array, size_to_test, "char");
 
         // make sure we don't falsely say the next kernel is correct because we've left the correct answer sitting in memory
         hipMemset(device_output_array, 0, size_to_test);
 
         // generate GPU uint output
         double elapsed_time_uint = doGPUShiftUInt(device_input_array,
                                    device_output_array, shift_amount, size_to_test, CUDA_BLOCK_SIZE);
         checkResults(text_host, device_output_array, size_to_test, "uint");
 
         // make sure we don't falsely say the next kernel is correct because we've left the correct answer sitting in memory
         hipMemset(device_output_array, 0, size_to_test);
 
         // generate GPU uint2 output
         double elapsed_time_uint2 = doGPUShiftUInt2(device_input_array,
                                     device_output_array, shift_amount, size_to_test, CUDA_BLOCK_SIZE);
         checkResults(text_host, device_output_array, size_to_test, "uint2");
 
         // make sure we don't falsely say the next kernel is correct because we've left the correct answer sitting in memory
         hipMemset(device_output_array, 0, size_to_test);
 
         std::cout << std::setw(15) << size_to_test / 1E6 << " " 
                   << std::setw(15) << 2 * size_to_test / (elapsed_time_char / 1000.) / 1E9 
                   << std::setw(15) << 2 * size_to_test / (elapsed_time_uint / 1000.) / 1E9
                   << std::setw(15) << 2 * size_to_test / (elapsed_time_uint2 / 1000.) / 1E9
                   << std::endl;
     }
 
     // deallocate memory
     hipFree(device_input_array);
     hipFree(device_output_array);
 
     return exit_code;
 }
 